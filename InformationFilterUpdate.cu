#include "hip/hip_runtime.h"
//
// File: InformationFilterUpdate.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 07-Sep-2020 09:07:32
//

// Include Files
#include "InformationFilterUpdate.h"
#include "MWCudaDimUtility.hpp"
#include <cmath>

// Function Declarations
static __global__ void InformationFilterUpdate_kernel1(double Rw[4]);
static __global__ void InformationFilterUpdate_kernel2(const double Rw[2],
  double b_Rw[4]);
static __global__ void InformationFilterUpdate_kernel3(const double r, const
  double t, const double Rw[4], double Rw_inv[4]);
static __global__ void InformationFilterUpdate_kernel4(const double r, const
  double t, const double Rw[4], double Rw_inv[4]);
static __global__ void InformationFilterUpdate_kernel5(double Fk_inv[16]);
static __global__ void InformationFilterUpdate_kernel6(const double T, double x
  [16]);
static __global__ void InformationFilterUpdate_kernel7(const signed char iv[4],
  const signed char iv1[4], signed char ipiv[4], double x[16]);
static __global__ void InformationFilterUpdate_kernel8(signed char p[4]);
static __global__ void InformationFilterUpdate_kernel9(const double T, double
  Gk[8]);
static __global__ void ab_InformationFilterUpdate_kern(const double r, const
  double delta, const double L_imuToRear, const double xk_m_out[4], double
  H_linear[52], double hk[13]);
static __global__ void b_InformationFilterUpdate_kerne(const double a[16], const
  double Fk_inv[16], const int i, double x[16]);
static __global__ void bb_InformationFilterUpdate_kern(const double H_linear[52],
  double A[52]);
static __global__ void c_InformationFilterUpdate_kerne(const double Fk_inv[16],
  const double x[16], const int i, double Ih[16]);
static __global__ void cb_InformationFilterUpdate_kern(const double xk_m_out[4],
  const double H_linear[52], const double hk[13], const double y_meas[13],
  double b_y_meas[13]);
static __global__ void d_InformationFilterUpdate_kerne(const double Ih[16],
  const double Gk[8], double b_Gk[8]);
static __global__ void db_InformationFilterUpdate_kern(const double y_meas[13],
  const double C[52], const double ik[4], double op[4]);
static __global__ void e_InformationFilterUpdate_kerne(const double Gk[8], const
  double b_Gk[8], const double Rw_inv[4], double x[4]);
static __global__ void f_InformationFilterUpdate_kerne(const double r, const
  double t, const double x[4], double Rw[4]);
static __global__ void g_InformationFilterUpdate_kerne(const double r, const
  double t, const double x[4], double Rw[4]);
static __global__ void h_InformationFilterUpdate_kerne(const double Gk[8], const
  double Ih[16], double b_Ih[8]);
static __global__ void i_InformationFilterUpdate_kerne(const double Rw[4], const
  double Ih[8], double b_Ih[8]);
static __global__ void j_InformationFilterUpdate_kerne(const double Gk[8], const
  double Ih[8], double calcEq[16]);
static __global__ void k_InformationFilterUpdate_kerne(const double
  initialization_vec[4], double ih[4]);
static __global__ void l_InformationFilterUpdate_kerne(const double ih[4], const
  double a[16], double ik[4]);
static __global__ void m_InformationFilterUpdate_kerne(const double ik[4], const
  double Fk_inv[16], double ih[4]);
static __global__ void n_InformationFilterUpdate_kerne(const double calcEq[16],
  const double ih[4], double ik[4]);
static __global__ void o_InformationFilterUpdate_kerne(double Fk_inv[16]);
static __global__ void p_InformationFilterUpdate_kerne(const double calcEq[16],
  const double Ih[16], double x[16]);
static __global__ void q_InformationFilterUpdate_kerne(signed char ipiv[4]);
static __global__ void r_InformationFilterUpdate_kerne(signed char p[4]);
static __global__ void s_InformationFilterUpdate_kerne(const double ik[4], const
  double Fk_inv[16], double xk_m_out[4]);
static __global__ void t_InformationFilterUpdate_kerne(const double
  B_usedMeas_vec[13], const double Re[13], double Re_inv[13]);
static __global__ void u_InformationFilterUpdate_kerne(double Re_inv[169]);
static __global__ void v_InformationFilterUpdate_kerne(const double Re_inv[13],
  double b_Re_inv[169]);
static __global__ void w_InformationFilterUpdate_kerne(double H_linear[52]);
static __global__ void x_InformationFilterUpdate_kerne(const signed char iv2[4],
  const signed char iv3[4], const signed char iv1[4], double H_linear[52]);
static __global__ void y_InformationFilterUpdate_kerne(const double xk_m_out[4],
  const double H_linear[52], double hk[13]);

// Function Definitions

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Rw[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel1
  (double Rw[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    // ---- Prediction step -------------------------
    Rw[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Rw[2]
//                double b_Rw[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel2(
  const double Rw[2], double b_Rw[4])
{
  unsigned int threadId;
  int j;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  j = static_cast<int>(threadId);
  if (j < 2) {
    b_Rw[j + (j << 1)] = Rw[j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double r
//                const double t
//                const double Rw[4]
//                double Rw_inv[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel3(
  const double r, const double t, const double Rw[4], double Rw_inv[4])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    Rw_inv[0] = Rw[3] / Rw[0] * t;
    Rw_inv[1] = -r * t;
    Rw_inv[2] = -Rw[2] / Rw[0] * t;
    Rw_inv[3] = t;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double r
//                const double t
//                const double Rw[4]
//                double Rw_inv[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel4(
  const double r, const double t, const double Rw[4], double Rw_inv[4])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    Rw_inv[0] = Rw[3] / Rw[1] * t;
    Rw_inv[1] = -t;
    Rw_inv[2] = -Rw[2] / Rw[1] * t;
    Rw_inv[3] = r * t;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Fk_inv[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel5
  (double Fk_inv[16])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 16) {
    //  System matrix
    Fk_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double T
//                double x[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel6(
  const double T, double x[16])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    x[0] = 1.0;
    x[4] = T;
    x[8] = 0.0;
    x[12] = 0.0;
    x[2] = 0.0;
    x[6] = 0.0;
    x[10] = 1.0;
    x[14] = T;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv[4]
//                const signed char iv1[4]
//                signed char ipiv[4]
//                double x[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel7(
  const signed char iv[4], const signed char iv1[4], signed char ipiv[4], double
  x[16])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    x[(i << 2) + 1] = static_cast<double>(iv1[i]);
    x[(i << 2) + 3] = static_cast<double>(iv[i]);
    ipiv[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel8
  (signed char p[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    p[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double T
//                double Gk[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel9(
  const double T, double Gk[8])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    //  Noise matrix
    Gk[0] = T * T / 2.0;
    Gk[4] = 0.0;
    Gk[1] = T;
    Gk[5] = 0.0;
    Gk[2] = 0.0;
    Gk[6] = T * T / 2.0;
    Gk[3] = 0.0;
    Gk[7] = T;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double r
//                const double delta
//                const double L_imuToRear
//                const double xk_m_out[4]
//                double H_linear[52]
//                double hk[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void ab_InformationFilterUpdate_kern(
  const double r, const double delta, const double L_imuToRear, const double
  xk_m_out[4], double H_linear[52], double hk[13])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    hk[0] -= xk_m_out[2] * xk_m_out[2] * L_imuToRear;
    hk[1] += xk_m_out[0] * xk_m_out[2];
    hk[10] = xk_m_out[0] * cos(delta) + r * xk_m_out[2] * sin(delta);
    H_linear[26] = -2.0 * xk_m_out[2] * L_imuToRear;
    H_linear[1] = xk_m_out[2];
    H_linear[27] = xk_m_out[0];
    H_linear[4] = cos(delta);
    H_linear[10] = cos(delta);
    H_linear[36] = r * sin(delta);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double a[16]
//                const double Fk_inv[16]
//                const int i
//                double x[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void b_InformationFilterUpdate_kerne(
  const double a[16], const double Fk_inv[16], const int i, double x[16])
{
  unsigned int threadId;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId);
  if (i5 < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Fk_inv[i4 + (i << 2)] * a[i4 + (i5 << 2)];
    }

    x[i + (i5 << 2)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double H_linear[52]
//                double A[52]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void bb_InformationFilterUpdate_kern(
  const double H_linear[52], double A[52])
{
  unsigned int threadId;
  int i;
  int i4;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i4 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i4)) / 4U);
  if (i < 13) {
    A[i4 + (i << 2)] = H_linear[i + 13 * i4];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Fk_inv[16]
//                const double x[16]
//                const int i
//                double Ih[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void c_InformationFilterUpdate_kerne(
  const double Fk_inv[16], const double x[16], const int i, double Ih[16])
{
  unsigned int threadId;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId);
  if (i5 < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += x[i + (i4 << 2)] * Fk_inv[i4 + (i5 << 2)];
    }

    Ih[i + (i5 << 2)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double xk_m_out[4]
//                const double H_linear[52]
//                const double hk[13]
//                const double y_meas[13]
//                double b_y_meas[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void cb_InformationFilterUpdate_kern(
  const double xk_m_out[4], const double H_linear[52], const double hk[13],
  const double y_meas[13], double b_y_meas[13])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 13) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += H_linear[i + 13 * i4] * xk_m_out[i4];
    }

    b_y_meas[i] = (y_meas[i] - hk[i]) + d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Ih[16]
//                const double Gk[8]
//                double b_Gk[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void d_InformationFilterUpdate_kerne(
  const double Ih[16], const double Gk[8], double b_Gk[8])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 4U);
  if (i < 2) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Gk[i4 + (i << 2)] * Ih[i4 + (i5 << 2)];
    }

    b_Gk[i + (i5 << 1)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double y_meas[13]
//                const double C[52]
//                const double ik[4]
//                double op[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void db_InformationFilterUpdate_kern(
  const double y_meas[13], const double C[52], const double ik[4], double op[4])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 13; i4++) {
      d += C[i + (i4 << 2)] * y_meas[i4];
    }

    op[i] = ik[i] + d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[8]
//                const double b_Gk[8]
//                const double Rw_inv[4]
//                double x[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void e_InformationFilterUpdate_kerne(
  const double Gk[8], const double b_Gk[8], const double Rw_inv[4], double x[4])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 2U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 2U);
  if (i < 2) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += b_Gk[i + (i4 << 1)] * Gk[i4 + (i5 << 2)];
    }

    x[i + (i5 << 1)] = d + Rw_inv[i + (i5 << 1)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double r
//                const double t
//                const double x[4]
//                double Rw[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void f_InformationFilterUpdate_kerne(
  const double r, const double t, const double x[4], double Rw[4])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    Rw[0] = x[3] / x[0] * t;
    Rw[1] = -r * t;
    Rw[2] = -x[2] / x[0] * t;
    Rw[3] = t;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double r
//                const double t
//                const double x[4]
//                double Rw[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void g_InformationFilterUpdate_kerne(
  const double r, const double t, const double x[4], double Rw[4])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    Rw[0] = x[3] / x[1] * t;
    Rw[1] = -t;
    Rw[2] = -x[2] / x[1] * t;
    Rw[3] = r * t;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[8]
//                const double Ih[16]
//                double b_Ih[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void h_InformationFilterUpdate_kerne(
  const double Gk[8], const double Ih[16], double b_Ih[8])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 2U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 2U);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Ih[i + (i4 << 2)] * Gk[i4 + (i5 << 2)];
    }

    b_Ih[i + (i5 << 2)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Rw[4]
//                const double Ih[8]
//                double b_Ih[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void i_InformationFilterUpdate_kerne(
  const double Rw[4], const double Ih[8], double b_Ih[8])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 2U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 2U);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 2; i4++) {
      d += Ih[i + (i4 << 2)] * Rw[i4 + (i5 << 1)];
    }

    b_Ih[i + (i5 << 2)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[8]
//                const double Ih[8]
//                double calcEq[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void j_InformationFilterUpdate_kerne(
  const double Gk[8], const double Ih[8], double calcEq[16])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 4U);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 2; i4++) {
      d += Ih[i + (i4 << 2)] * Gk[i5 + (i4 << 2)];
    }

    calcEq[i + (i5 << 2)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double initialization_vec[4]
//                double ih[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void k_InformationFilterUpdate_kerne(
  const double initialization_vec[4], double ih[4])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    ih[0] = initialization_vec[2];
    ih[1] = 0.0;
    ih[2] = 0.0;
    ih[3] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ih[4]
//                const double a[16]
//                double ik[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void l_InformationFilterUpdate_kerne(
  const double ih[4], const double a[16], double ik[4])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += a[i + (i4 << 2)] * ih[i4];
    }

    ik[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ik[4]
//                const double Fk_inv[16]
//                double ih[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void m_InformationFilterUpdate_kerne(
  const double ik[4], const double Fk_inv[16], double ih[4])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Fk_inv[i4 + (i << 2)] * ik[i4];
    }

    ih[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double calcEq[16]
//                const double ih[4]
//                double ik[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void n_InformationFilterUpdate_kerne(
  const double calcEq[16], const double ih[4], double ik[4])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += calcEq[i + (i4 << 2)] * ih[i4];
    }

    ik[i] = ih[i] - d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Fk_inv[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void o_InformationFilterUpdate_kerne
  (double Fk_inv[16])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 16) {
    Fk_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double calcEq[16]
//                const double Ih[16]
//                double x[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void p_InformationFilterUpdate_kerne(
  const double calcEq[16], const double Ih[16], double x[16])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 4U);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += calcEq[i + (i4 << 2)] * Ih[i4 + (i5 << 2)];
    }

    x[i + (i5 << 2)] = Ih[i + (i5 << 2)] - d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void q_InformationFilterUpdate_kerne
  (signed char ipiv[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    ipiv[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void r_InformationFilterUpdate_kerne
  (signed char p[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    p[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ik[4]
//                const double Fk_inv[16]
//                double xk_m_out[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void s_InformationFilterUpdate_kerne(
  const double ik[4], const double Fk_inv[16], double xk_m_out[4])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Fk_inv[i + (i4 << 2)] * ik[i4];
    }

    xk_m_out[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double B_usedMeas_vec[13]
//                const double Re[13]
//                double Re_inv[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void t_InformationFilterUpdate_kerne(
  const double B_usedMeas_vec[13], const double Re[13], double Re_inv[13])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 13) {
    Re_inv[i] = 1.0 / Re[i] * B_usedMeas_vec[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Re_inv[169]
// Return Type  : void
//
static __global__ __launch_bounds__(192, 1) void u_InformationFilterUpdate_kerne
  (double Re_inv[169])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 169) {
    Re_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Re_inv[13]
//                double b_Re_inv[169]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void v_InformationFilterUpdate_kerne(
  const double Re_inv[13], double b_Re_inv[169])
{
  unsigned int threadId;
  int j;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  j = static_cast<int>(threadId);
  if (j < 13) {
    b_Re_inv[j + 13 * j] = Re_inv[j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double H_linear[52]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void w_InformationFilterUpdate_kerne
  (double H_linear[52])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    H_linear[48] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv2[4]
//                const signed char iv3[4]
//                const signed char iv1[4]
//                double H_linear[52]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void x_InformationFilterUpdate_kerne(
  const signed char iv2[4], const signed char iv3[4], const signed char iv1[4],
  double H_linear[52])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    H_linear[13 * i] = static_cast<double>(iv1[i]);
    H_linear[13 * i + 2] = static_cast<double>(iv3[i]);
    H_linear[13 * i + 3] = static_cast<double>(iv2[i]);
    H_linear[13 * i + 4] = 0.0;
    H_linear[13 * i + 5] = 0.0;
    H_linear[13 * i + 10] = 0.0;
    H_linear[13 * i + 11] = static_cast<double>(iv2[i]);
    H_linear[13 * i + 12] = static_cast<double>(iv2[i]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double xk_m_out[4]
//                const double H_linear[52]
//                double hk[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void y_InformationFilterUpdate_kerne(
  const double xk_m_out[4], const double H_linear[52], double hk[13])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 13) {
    //  Nonlinear parts
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += H_linear[i + 13 * i4] * xk_m_out[i4];
    }

    hk[i] = d;
  }
}

//
// Arguments    : const double y_meas[13]
//                const double B_usedMeas_vec[13]
//                const double initialization_vec[4]
//                double delta
//                const double Rw[2]
//                const double Re[13]
//                double L_imuToRear
//                double L_geometricWheelbase
//                const double L_trackWidth[5]
//                const double L_axlePos[5]
//                double T
//                double xk_m_out[4]
//                double op[4]
// Return Type  : void
//
void InformationFilterUpdate(const double y_meas[13], const double
  B_usedMeas_vec[13], const double initialization_vec[4], double delta, const
  double Rw[2], const double Re[13], double L_imuToRear, double
  L_geometricWheelbase, const double L_trackWidth[5], const double L_axlePos[5],
  double T, double xk_m_out[4], double op[4])
{
  int j;
  double r;
  double t;
  static const signed char iv[4] = { 0, 0, 0, 1 };

  int c;
  static const signed char iv1[4] = { 0, 1, 0, 0 };

  int ar;
  int iy;
  int k;
  int ix;
  signed char i1;
  int i2;
  int ia;
  int jy;
  int i;
  int kAcol;
  int i3;
  static const double a[16] = { 100.0, 0.0, 0.0, 0.0, 0.0, 33.333333333333336,
    0.0, 0.0, 0.0, 0.0, 10000.0, 0.0, 0.0, 0.0, 0.0, 500.0 };

  static const signed char iv2[4] = { 1, 0, 0, 0 };

  static const signed char iv3[4] = { 0, 0, 1, 0 };

  double (*gpu_Rw)[4];
  double (*b_gpu_Rw)[2];
  double (*gpu_Rw_inv)[4];
  double (*gpu_Fk_inv)[16];
  double (*gpu_x)[16];
  signed char (*gpu_iv)[4];
  signed char (*gpu_iv1)[4];
  signed char (*gpu_ipiv)[4];
  signed char (*gpu_p)[4];
  double (*gpu_Gk)[8];
  double (*gpu_a)[16];
  double (*gpu_Ih)[16];
  double (*b_gpu_Gk)[8];
  double (*b_gpu_x)[4];
  double (*b_gpu_Ih)[8];
  double (*c_gpu_Ih)[8];
  double (*gpu_calcEq)[16];
  double (*gpu_initialization_vec)[4];
  double (*gpu_ih)[4];
  double (*gpu_ik)[4];
  double (*gpu_xk_m_out)[4];
  double (*gpu_B_usedMeas_vec)[13];
  double (*gpu_Re)[13];
  double (*gpu_Re_inv)[13];
  double (*b_gpu_Re_inv)[169];
  double (*gpu_H_linear)[52];
  signed char (*gpu_iv2)[4];
  signed char (*gpu_iv3)[4];
  double (*gpu_hk)[13];
  double (*gpu_A)[52];
  double (*gpu_y_meas)[13];
  double (*b_gpu_y_meas)[13];
  double (*gpu_C)[52];
  double (*gpu_op)[4];
  boolean_T syncIsDirty;
  hipMallocManaged(&gpu_C, 416ULL);
  hipMallocManaged(&b_gpu_y_meas, 104ULL);
  hipMallocManaged(&gpu_A, 416ULL);
  hipMallocManaged(&gpu_hk, 104ULL);
  hipMallocManaged(&gpu_H_linear, 416ULL);
  hipMallocManaged(&b_gpu_Re_inv, 1352ULL);
  hipMallocManaged(&gpu_Re_inv, 104ULL);
  hipMallocManaged(&gpu_ik, 32ULL);
  hipMallocManaged(&gpu_ih, 32ULL);
  hipMallocManaged(&gpu_calcEq, 128ULL);
  hipMallocManaged(&c_gpu_Ih, 64ULL);
  hipMallocManaged(&b_gpu_Ih, 64ULL);
  hipMallocManaged(&b_gpu_x, 32ULL);
  hipMallocManaged(&b_gpu_Gk, 64ULL);
  hipMallocManaged(&gpu_Ih, 128ULL);
  hipMallocManaged(&gpu_Gk, 64ULL);
  hipMallocManaged(&gpu_p, 4ULL);
  hipMallocManaged(&gpu_ipiv, 4ULL);
  hipMallocManaged(&gpu_x, 128ULL);
  hipMallocManaged(&gpu_Fk_inv, 128ULL);
  hipMallocManaged(&gpu_Rw_inv, 32ULL);
  hipMallocManaged(&gpu_Rw, 32ULL);
  hipMallocManaged(&gpu_op, 32ULL);
  hipMallocManaged(&gpu_xk_m_out, 32ULL);
  hipMallocManaged(&b_gpu_Rw, 16ULL);
  hipMallocManaged(&gpu_iv, 4ULL);
  hipMallocManaged(&gpu_iv1, 4ULL);
  hipMallocManaged(&gpu_a, 128ULL);
  hipMallocManaged(&gpu_initialization_vec, 32ULL);
  hipMallocManaged(&gpu_B_usedMeas_vec, 104ULL);
  hipMallocManaged(&gpu_Re, 104ULL);
  hipMallocManaged(&gpu_iv2, 4ULL);
  hipMallocManaged(&gpu_iv3, 4ULL);
  hipMallocManaged(&gpu_y_meas, 104ULL);
  hipMemcpy(gpu_y_meas, (void *)&y_meas[0], 104ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv3, (void *)&iv3[0], 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv2, (void *)&iv2[0], 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_Re, (void *)&Re[0], 104ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_B_usedMeas_vec, (void *)&B_usedMeas_vec[0], 104ULL,
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_initialization_vec, (void *)&initialization_vec[0], 32ULL,
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_a, (void *)&a[0], 128ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv1, (void *)&iv1[0], 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv, (void *)&iv[0], 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(b_gpu_Rw, (void *)&Rw[0], 16ULL, hipMemcpyHostToDevice);

  // ---- Prediction step -------------------------
  InformationFilterUpdate_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Rw);
  InformationFilterUpdate_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_Rw, *gpu_Rw);
  hipDeviceSynchronize();
  if (std::abs((*gpu_Rw)[1]) > std::abs((*gpu_Rw)[0])) {
    r = (*gpu_Rw)[0] / (*gpu_Rw)[1];
    InformationFilterUpdate_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(r,
      1.0 / (r * (*gpu_Rw)[3] - (*gpu_Rw)[2]), *gpu_Rw, *gpu_Rw_inv);
  } else {
    r = (*gpu_Rw)[1] / (*gpu_Rw)[0];
    InformationFilterUpdate_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(r,
      1.0 / ((*gpu_Rw)[3] - r * (*gpu_Rw)[2]), *gpu_Rw, *gpu_Rw_inv);
  }

  //  System matrix
  InformationFilterUpdate_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Fk_inv);
  InformationFilterUpdate_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(T,
    *gpu_x);
  InformationFilterUpdate_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_iv, *gpu_iv1, *gpu_ipiv, *gpu_x);
  syncIsDirty = true;
  for (j = 0; j < 3; j++) {
    c = j * 5;
    ar = 2 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    r = std::abs((*gpu_x)[c]);
    for (k = 0; k <= ar; k++) {
      ix++;
      t = std::abs((*gpu_x)[ix]);
      if (t > r) {
        iy = k + 1;
        r = t;
      }
    }

    if ((*gpu_x)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        ar = iy;
        for (k = 0; k < 4; k++) {
          ix = j + k * 4;
          iy = ar + k * 4;
          r = (*gpu_x)[ix];
          (*gpu_x)[ix] = (*gpu_x)[iy];
          (*gpu_x)[iy] = r;
        }
      }

      i2 = (c - j) + 2;
      for (i = 0; i <= i2 - c; i++) {
        iy = (c + i) + 1;
        (*gpu_x)[iy] /= (*gpu_x)[c];
      }
    }

    ar = 2 - j;
    iy = c + 6;
    jy = c + 4;
    for (ia = 0; ia <= ar; ia++) {
      r = (*gpu_x)[jy];
      if ((*gpu_x)[jy] != 0.0) {
        ix = c;
        i2 = iy - 2;
        i3 = iy - j;
        for (kAcol = 0; kAcol <= i3 - i2; kAcol++) {
          i = (iy + kAcol) - 1;
          (*gpu_x)[i] += (*gpu_x)[ix + 1] * -r;
          ix++;
        }
      }

      jy += 4;
      iy += 4;
    }
  }

  InformationFilterUpdate_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_p);
  syncIsDirty = true;
  for (k = 0; k < 3; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*gpu_ipiv)[k] > k + 1) {
      iy = (*gpu_p)[(*gpu_ipiv)[k] - 1];
      (*gpu_p)[(*gpu_ipiv)[k] - 1] = (*gpu_p)[k];
      (*gpu_p)[k] = static_cast<signed char>(iy);
    }
  }

  for (k = 0; k < 4; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    i1 = (*gpu_p)[k];
    (*gpu_Fk_inv)[k + (((*gpu_p)[k] - 1) << 2)] = 1.0;
    for (j = 0; j <= 3 - k; j++) {
      ia = k + j;
      if ((*gpu_Fk_inv)[ia + ((i1 - 1) << 2)] != 0.0) {
        for (i = 0; i <= 2 - ia; i++) {
          iy = (ia + i) + 1;
          (*gpu_Fk_inv)[iy + ((i1 - 1) << 2)] -= (*gpu_Fk_inv)[ia + ((i1 - 1) <<
            2)] * (*gpu_x)[iy + (ia << 2)];
        }
      }
    }
  }

  for (j = 0; j < 4; j++) {
    iy = (j << 2) - 1;
    for (k = 0; k < 4; k++) {
      jy = 4 - k;
      kAcol = (3 - k) << 2;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Fk_inv)[(iy - k) + 4] != 0.0) {
        (*gpu_Fk_inv)[(iy - k) + 4] /= (*gpu_x)[(kAcol - k) + 3];
        for (i = 0; i <= jy - 2; i++) {
          (*gpu_Fk_inv)[(i + iy) + 1] -= (*gpu_Fk_inv)[(iy - k) + 4] * (*gpu_x)
            [i + kAcol];
        }
      }
    }
  }

  //  Noise matrix
  InformationFilterUpdate_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(T,
    *gpu_Gk);

  //  Prediction step alternative 2. Gives easier matrix to invert
  for (i = 0; i < 4; i++) {
    b_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_a, *gpu_Fk_inv, i, *gpu_x);
    c_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_Fk_inv, *gpu_x, i, *gpu_Ih);
  }

  d_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Ih, *gpu_Gk, *b_gpu_Gk);
  e_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Gk, *b_gpu_Gk, *gpu_Rw_inv, *b_gpu_x);
  hipDeviceSynchronize();
  if (std::abs((*b_gpu_x)[1]) > std::abs((*b_gpu_x)[0])) {
    r = (*b_gpu_x)[0] / (*b_gpu_x)[1];
    g_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(r,
      1.0 / (r * (*b_gpu_x)[3] - (*b_gpu_x)[2]), *b_gpu_x, *gpu_Rw);
  } else {
    r = (*b_gpu_x)[1] / (*b_gpu_x)[0];
    f_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(r,
      1.0 / ((*b_gpu_x)[3] - r * (*b_gpu_x)[2]), *b_gpu_x, *gpu_Rw);
  }

  h_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Gk, *gpu_Ih, *b_gpu_Ih);
  i_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Rw, *b_gpu_Ih, *c_gpu_Ih);
  j_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Gk, *c_gpu_Ih, *gpu_calcEq);
  k_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_initialization_vec, *gpu_ih);
  l_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih, *gpu_a, *gpu_ik);
  m_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ik, *gpu_Fk_inv, *gpu_ih);
  n_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_calcEq, *gpu_ih, *gpu_ik);
  o_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Fk_inv);
  p_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_calcEq, *gpu_Ih, *gpu_x);
  q_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 3; j++) {
    c = j * 5;
    ar = 2 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    r = std::abs((*gpu_x)[c]);
    for (k = 0; k <= ar; k++) {
      ix++;
      t = std::abs((*gpu_x)[ix]);
      if (t > r) {
        iy = k + 1;
        r = t;
      }
    }

    if ((*gpu_x)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        ar = iy;
        for (k = 0; k < 4; k++) {
          ix = j + k * 4;
          iy = ar + k * 4;
          r = (*gpu_x)[ix];
          (*gpu_x)[ix] = (*gpu_x)[iy];
          (*gpu_x)[iy] = r;
        }
      }

      i2 = (c - j) + 2;
      for (i = 0; i <= i2 - c; i++) {
        iy = (c + i) + 1;
        (*gpu_x)[iy] /= (*gpu_x)[c];
      }
    }

    ar = 2 - j;
    iy = c + 6;
    jy = c + 4;
    for (ia = 0; ia <= ar; ia++) {
      r = (*gpu_x)[jy];
      if ((*gpu_x)[jy] != 0.0) {
        ix = c;
        i2 = iy - 2;
        i3 = iy - j;
        for (kAcol = 0; kAcol <= i3 - i2; kAcol++) {
          i = (iy + kAcol) - 1;
          (*gpu_x)[i] += (*gpu_x)[ix + 1] * -r;
          ix++;
        }
      }

      jy += 4;
      iy += 4;
    }
  }

  r_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_p);
  syncIsDirty = true;
  for (k = 0; k < 3; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*gpu_ipiv)[k] > k + 1) {
      iy = (*gpu_p)[(*gpu_ipiv)[k] - 1];
      (*gpu_p)[(*gpu_ipiv)[k] - 1] = (*gpu_p)[k];
      (*gpu_p)[k] = static_cast<signed char>(iy);
    }
  }

  for (k = 0; k < 4; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    i1 = (*gpu_p)[k];
    (*gpu_Fk_inv)[k + (((*gpu_p)[k] - 1) << 2)] = 1.0;
    for (j = 0; j <= 3 - k; j++) {
      ia = k + j;
      if ((*gpu_Fk_inv)[ia + ((i1 - 1) << 2)] != 0.0) {
        for (i = 0; i <= 2 - ia; i++) {
          iy = (ia + i) + 1;
          (*gpu_Fk_inv)[iy + ((i1 - 1) << 2)] -= (*gpu_Fk_inv)[ia + ((i1 - 1) <<
            2)] * (*gpu_x)[iy + (ia << 2)];
        }
      }
    }
  }

  for (j = 0; j < 4; j++) {
    iy = (j << 2) - 1;
    for (k = 0; k < 4; k++) {
      jy = 4 - k;
      kAcol = (3 - k) << 2;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Fk_inv)[(iy - k) + 4] != 0.0) {
        (*gpu_Fk_inv)[(iy - k) + 4] /= (*gpu_x)[(kAcol - k) + 3];
        for (i = 0; i <= jy - 2; i++) {
          (*gpu_Fk_inv)[(i + iy) + 1] -= (*gpu_Fk_inv)[(iy - k) + 4] * (*gpu_x)
            [i + kAcol];
        }
      }
    }
  }

  s_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ik, *gpu_Fk_inv, *gpu_xk_m_out);

  // ---- Measurement step --------------------------
  //  Extract front and rear track widths
  //  drive axle 1
  //  drive axle 2
  r = L_axlePos[0] + L_geometricWheelbase;
  t_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_B_usedMeas_vec, *gpu_Re, *gpu_Re_inv);
  u_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(192U, 1U, 1U)>>>
    (*b_gpu_Re_inv);
  v_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Re_inv, *b_gpu_Re_inv);

  //  Linear parts
  hipDeviceSynchronize();
  (*gpu_H_linear)[1] = 0.0;
  (*gpu_H_linear)[14] = 0.0;
  (*gpu_H_linear)[27] = 0.0;
  (*gpu_H_linear)[40] = L_imuToRear;
  (*gpu_H_linear)[6] = 1.0;
  (*gpu_H_linear)[19] = 0.0;
  (*gpu_H_linear)[32] = -L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[45] = 0.0;
  (*gpu_H_linear)[7] = 1.0;
  (*gpu_H_linear)[20] = 0.0;
  (*gpu_H_linear)[33] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[46] = 0.0;
  (*gpu_H_linear)[8] = 1.0;
  (*gpu_H_linear)[21] = 0.0;
  (*gpu_H_linear)[34] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[47] = 0.0;
  (*gpu_H_linear)[9] = 1.0;
  (*gpu_H_linear)[22] = 0.0;
  (*gpu_H_linear)[35] = L_trackWidth[3] / 2.0;
  w_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_H_linear);
  x_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_iv2, *gpu_iv3, *gpu_iv1, *gpu_H_linear);

  //  Nonlinear parts
  y_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_xk_m_out, *gpu_H_linear, *gpu_hk);
  hipDeviceSynchronize();
  (*gpu_hk)[4] = ((*gpu_xk_m_out)[0] - L_trackWidth[0] * (*gpu_xk_m_out)[2] /
                  2.0) * std::cos(delta) + r * (*gpu_xk_m_out)[2] * std::sin
    (delta);
  (*gpu_hk)[5] = ((*gpu_xk_m_out)[0] + L_trackWidth[0] * (*gpu_xk_m_out)[2] /
                  2.0) * std::cos(delta) + r * (*gpu_xk_m_out)[2] * std::sin
    (delta);
  (*gpu_H_linear)[30] = -L_trackWidth[0] * std::cos(delta) / 2.0 + r * std::sin
    (delta);
  (*gpu_H_linear)[5] = std::cos(delta);
  (*gpu_H_linear)[31] = L_trackWidth[0] * std::cos(delta) / 2.0 + r * std::sin
    (delta);
  ab_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(r,
    delta, L_imuToRear, *gpu_xk_m_out, *gpu_H_linear, *gpu_hk);
  bb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_H_linear, *gpu_A);
  syncIsDirty = true;
  for (iy = 0; iy < 13; iy++) {
    kAcol = iy << 2;
    i2 = kAcol - 3;
    for (i = 0; i <= kAcol - i2; i++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_C)[kAcol + i] = 0.0;
    }
  }

  for (iy = 0; iy < 13; iy++) {
    jy = iy * 13 + 1;
    kAcol = iy << 2;
    ar = -1;
    i2 = jy - 12;
    for (ix = 0; ix <= jy - i2; ix++) {
      k = jy + ix;
      ia = ar;
      i3 = kAcol - 2;
      c = kAcol + 1;
      for (i = 0; i <= c - i3; i++) {
        j = kAcol + i;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_C)[j] += (*b_gpu_Re_inv)[k - 1] * (*gpu_A)[ia];
      }

      ar += 4;
    }
  }

  cb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_xk_m_out, *gpu_H_linear, *gpu_hk, *gpu_y_meas, *b_gpu_y_meas);
  db_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_y_meas, *gpu_C, *gpu_ik, *gpu_op);
  hipDeviceSynchronize();
  hipMemcpy(&xk_m_out[0], gpu_xk_m_out, 32ULL, hipMemcpyDeviceToHost);
  hipMemcpy(&op[0], gpu_op, 32ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_y_meas);
  hipFree(*gpu_iv3);
  hipFree(*gpu_iv2);
  hipFree(*gpu_Re);
  hipFree(*gpu_B_usedMeas_vec);
  hipFree(*gpu_initialization_vec);
  hipFree(*gpu_a);
  hipFree(*gpu_iv1);
  hipFree(*gpu_iv);
  hipFree(*b_gpu_Rw);
  hipFree(*gpu_xk_m_out);
  hipFree(*gpu_op);
  hipFree(*gpu_Rw);
  hipFree(*gpu_Rw_inv);
  hipFree(*gpu_Fk_inv);
  hipFree(*gpu_x);
  hipFree(*gpu_ipiv);
  hipFree(*gpu_p);
  hipFree(*gpu_Gk);
  hipFree(*gpu_Ih);
  hipFree(*b_gpu_Gk);
  hipFree(*b_gpu_x);
  hipFree(*b_gpu_Ih);
  hipFree(*c_gpu_Ih);
  hipFree(*gpu_calcEq);
  hipFree(*gpu_ih);
  hipFree(*gpu_ik);
  hipFree(*gpu_Re_inv);
  hipFree(*b_gpu_Re_inv);
  hipFree(*gpu_H_linear);
  hipFree(*gpu_hk);
  hipFree(*gpu_A);
  hipFree(*b_gpu_y_meas);
  hipFree(*gpu_C);
}

//
// Arguments    : void
// Return Type  : void
//
void InformationFilterUpdate_initialize()
{
}

//
// Arguments    : void
// Return Type  : void
//
void InformationFilterUpdate_terminate()
{
  // (no terminate code required)
}

//
// File trailer for InformationFilterUpdate.cu
//
// [EOF]
//
